#pragma once
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

template<unsigned int k>
class BitSequence
{
public:
	__host__ BitSequence()
	{ 
	}
	__host__ BitSequence(char array[])
	{
		hipMemcpy(this->array, array, arSize, hipMemcpyHostToHost);
	}
	__host__ __device__ inline char GetBit(unsigned int index)
	{
		return array[index / 8] >> (index % 8) & 1;
	}
	__host__ __device__ inline void SetBit(unsigned int index, char value)
	{
		array[index / 8] = (array[index / 8] & (~(1 << (index % 8)))) | ((!!value) << (index % 8));
	}
	__host__ __device__ inline unsigned int *GetWord32(unsigned int word_index)
	{
		return (unsigned int*)(array + word_index * 32 / 8);
	}
	__host__ __device__ inline unsigned long long *GetWord64(unsigned int word_index)
	{
		return (unsigned long long*)(array + word_index * 64 / 8);
	}
	static const unsigned int arSize = (k + 7) / 8;
private:
	char array[arSize];
};

/*void f()
{
	BitSequence<1000> bs;
	BitSequence<1000000> bs2;
	bs.GetBit(0);
	bs.SetBit(0, 0);
	bs.GetWord32(0);
	bs.GetWord64(0);
	bs2.GetBit(0);
	bs2.GetWord32(0);
	bs2.GetWord64(0);
	bs2.SetBit(0, 0);
}*/