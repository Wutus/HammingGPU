#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>

using namespace std;

//#define ITER_GPU

#define CHECK_ERRORS(status) do{\
	if(hipSuccess != status) {\
		fprintf(stderr, "Cuda Error in %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(status));\
	}\
}while(0)

#define CHECK_ERRORS_FORMAT(status, format, ...) do{\
	if(hipSuccess != status) {\
		fprintf(stderr, "Cuda Error in %s:%d - %s", __FILE__, __LINE__, hipGetErrorString(status));\
		fprintf(stderr, format, __VA_ARGS__);\
		fprintf(stderr, "\n");\
	}\
}while(0)

#define K 10000 //Number of bits in one sequence
#define N 100000 //Number of sequences
#define L ((N*(N - 1)) / 2) //Number of comparisons
#define B 100000 //Number of maximum blocks per call

template<unsigned long long k>
class BitSequence;
class CudaTimer;

__global__ void checkSequencesGPU(BitSequence<K> * d_sequence, BitSequence<L> *d_odata, unsigned long long offset = 0);
__host__ __device__ char compareSequences(BitSequence<K> * sequence1, BitSequence<K> * sequence2);
__host__ __device__ void k2ij(unsigned long long k, unsigned int * i, unsigned int  * j);
__host__ __device__ unsigned long long ij2k(unsigned int i, unsigned int j);
void checkSequencesCPU(BitSequence<K> * sequence, BitSequence<L> * odata);
void PrintComparison(const BitSequence<K> & gpu_sequence, const BitSequence<K> & cpu_sequence);
bool ComparePairs(const vector<pair<int, int> > & gpu_result, const vector<pair<int, int> > & cpu_result);

ostream & operator<<(ostream & out, BitSequence<K> & sequence);
BitSequence<K> * Generate();
vector<pair<int, int>> ToPairVector(const BitSequence<L> & result_sequence);
void printAsMatrix(const BitSequence<L> & sequence, ostream & stream);

vector<pair<int, int> > findPairsGPU(BitSequence<K> * h_sequence);
vector<pair<int, int> > findPairsCPU(BitSequence<K> * sequence);

int main()
{
	hipError_t cudaStatus;
	printf("Starting sequence generation...\n");
	BitSequence<K>* sequence = Generate();
	printf("Ended sequence generation!\n");

	auto gpuRes = findPairsGPU(sequence);
	auto cpuRes = findPairsCPU(sequence);
	ComparePairs(gpuRes, cpuRes);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

template<unsigned long long k>
class BitSequence
{
public:
	__host__ BitSequence()
	{
	}
	__host__ BitSequence(char array[])
	{
		hipMemcpy(this->array, array, arSize*8, hipMemcpyHostToHost);
	}
	__host__ __device__ inline char GetBit(unsigned long long index) const
	{
		return (array[index / 64] >> (index % 64)) & 1;
	}
	__host__ __device__ inline void SetBit(unsigned long long index, char value)
	{
		array[index / 64] = (array[index / 64] & (~(1ull << (index % 64)))) | ((!!value) << (index % 64));
	}
	__host__ __device__ inline unsigned int *GetWord32(unsigned long long word_index)
	{
		return ((unsigned int*)array) + word_index);
	}
	__host__ __device__ inline unsigned long long *GetWord64(unsigned long long word_index)
	{
		return (array + word_index);
	}
	static const unsigned long long arSize = (k + 63) / 64;
private:
	unsigned long long array[arSize];
};

__host__ __device__ char compareSequences(BitSequence<K> * sequence1, BitSequence<K> * sequence2)
{
	int diff = 0;
	for (int j = 0; j < (K + 63) / 64; ++j)
	{
		unsigned long long int a, b, xor;
		a = *(sequence1->GetWord64(j));
		b = *(sequence2->GetWord64(j));
		xor = a ^ b;
		diff += xor == 0 ? 0 : (xor & (xor -1) ? 2 : 1);
		if (diff > 1)
		{
			return 0;
		}
	}
	return !!diff;
}

__host__ __device__ void k2ij(unsigned long long k, unsigned int * i, unsigned int  * j)
{
	*i = (unsigned int)ceil((0.5 * (-1 + sqrtl(1 + 8 * (k + 1)))));
	*j = (unsigned int)((k + 1) - 0.5 * (*i) * ((*i) - 1)) - 1;
}

__host__ __device__ unsigned long long ij2k(unsigned int i, unsigned int j)
{
	return ((unsigned long long)i) * (i - 1) / 2 + j;
}

void checkSequencesCPU(BitSequence<K> * sequence, BitSequence<L> * odata)
{
	unsigned long long numberOfComparisons = L;
	for (unsigned long long k = 0; k < numberOfComparisons; k += 32)
	{
		unsigned int result = 0;
		for (int i = 0; i < 32; i++)
		{
			unsigned int i1, i2;
			k2ij(k + i, &i1, &i2);
			result |= (unsigned int)(compareSequences(sequence + i1, sequence + i2)) << i;
		}
		*(odata->GetWord32(k / 32)) = result;
	}
}

class CudaTimer
{
public:
	CudaTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		started = false;
	}

	~CudaTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		started = true;
		hipEventRecord(start);
		hipEventSynchronize(start);
	}

	float Stop()
	{
		if (!started)
			return -1.0f;
		float ms;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		started = false;
		return ms;
	}
private:
	bool started;
	hipEvent_t start, stop;
};

void PrintComparison(const BitSequence<K> & gpu_sequence, const BitSequence<K> & cpu_sequence)
{
	for (unsigned long long i = 0; i < N*(N - 1) / 2; ++i)
	{
		if (cpu_sequence.GetBit(i) != gpu_sequence.GetBit(i))
		{
			unsigned int i1, i2;
			k2ij(i, &i1, &i2);
			cout << "Difference on comparison number " << i << " (" << i1 << ", " << i2 << ") GPU " << (short int)gpu_sequence.GetBit(i) << " CPU " << (short int)cpu_sequence.GetBit(i) << endl;
		}
	}
}

bool ComparePairs(const vector<pair<int, int> > & gpu_result, const vector<pair<int, int> > & cpu_result)
{
	unsigned long long gsize = gpu_result.size(), csize = cpu_result.size();
	unsigned long long n = gsize < csize ? gsize : csize;
	const vector<pair<int, int> > & gv = gsize < csize ? cpu_result : gpu_result;
	bool equal = true;

	if (gsize != csize)
	{
		cout << "Number of elements is not equal (GPU: " << gsize << ", CPU: " << csize << ") !" << endl;
		equal = false;
	}

	int i;
	for (i = 0; i < n; ++i)
	{
		if (gpu_result[i] != cpu_result[i])
		{
			cout << "Difference on " << i << ": GPU: (" << gpu_result[i].first << ", " << gpu_result[i].second << ") CPU: ("
				<< cpu_result[i].first << ", " << cpu_result[i].second << ")";
			equal = false;
		}
	}
	if (csize != gsize)
	{
		cout << "Rest pairs on " << ((csize < gsize) ? "CPU" : "GPU") << ":" << endl;
		for (; i < gv.size(); ++i)
		{
			cout << "(" << gv[i].first << ", " << gv[i].second << ")" << endl;
		}
	}
	if (equal)
	{
		cout << "Results are the same" << endl;
	}
	return equal;
}

__global__ void checkSequencesGPU(BitSequence<K> * d_sequence, BitSequence<L> *d_odata, unsigned long long offset = 0)
{
	unsigned long long i = threadIdx.x + blockIdx.x * blockDim.x + offset;
	unsigned int i1, i2;
	k2ij(i, &i1, &i2);
	i2 = compareSequences(d_sequence + i1, d_sequence + i2);
	i1 = __ballot(i2);
	*(d_odata->GetWord32(i / 32)) = i1;
}

ostream & operator<<(ostream & out, BitSequence<K> & sequence)
{
	for (unsigned long long i = 0; i < K; ++i)
	{
		out << (short int)sequence.GetBit(i);
	}
	return out;
}

BitSequence<K> * Generate()
{
	srand(2018);

	BitSequence<K> * r = new BitSequence<K>[N];

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < K / 32; j++)
		{
			*(r[i].GetWord32(j)) = rand() + rand()*RAND_MAX;
		}
	}
	return r;
}

vector<pair<int, int>> ToPairVector(const BitSequence<L> & result_sequence)
{
	vector<pair<int, int>> result;
	for (unsigned long long k = 0; k < L; k++)
	{
		if (result_sequence.GetBit(k))
		{
			unsigned int i, j;
			k2ij(k, &i, &j);
			result.push_back(make_pair(i, j));
		}
	}

	return result;
}

void printAsMatrix(const BitSequence<L> & sequence, ostream & stream)
{
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (j <= i)
			{
				cout << "  ";
			}
			else
			{
				cout << (short int)sequence.GetBit(ij2k(i, j)) << " ";
			}
		}
		cout << endl;
	}
}

vector<pair<int, int> > findPairsGPU(BitSequence<K> * h_sequence)
{
	BitSequence<K> *d_idata;
	BitSequence<L> *h_odata, *d_odata;
	CudaTimer timerCall, timerMemory;
	float xtime, xmtime;
	h_odata = new BitSequence<L>;
	unsigned long long inputSize = sizeof(BitSequence<K>)* N, outputSize = sizeof(BitSequence<L>);
	timerMemory.Start();
	CHECK_ERRORS(hipMalloc(&d_idata, inputSize));
	CHECK_ERRORS(hipMalloc(&d_odata, outputSize));
	CHECK_ERRORS(hipMemcpy(d_idata, h_sequence, inputSize, hipMemcpyHostToDevice));
	CHECK_ERRORS(hipMemcpy(d_odata, h_odata, outputSize, hipMemcpyHostToDevice));
	timerCall.Start();
	unsigned long long offset = 0;
#ifdef ITER_GPU
	for (; offset + B * 1024 < L; offset += B * 1024)
	{
		checkSequencesGPU << < B, 1024 >> > (d_idata, d_odata, offset);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if (L - offset >= 1024)
	{
		checkSequencesGPU << < (int)((L - offset) / 1024), 1024 >> > (d_idata, d_odata, offset);
		offset += (L - offset) * 1024;
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if ((L - offset) % 1024)
	{
		checkSequencesGPU << < 1, (int)(L - offset) >> > (d_idata, d_odata, offset);
		offset += L - offset;
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	CHECK_ERRORS(hipDeviceSynchronize());
#else
	if (L >= 1024)
	{
		checkSequencesGPU <<< (int)(L / 1024), 1024 >>> (d_idata, d_odata, 0);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if (L % 1024)
	{
		checkSequencesGPU <<< 1, L % 1024 >>> (d_idata, d_odata, (L / 1024) * 1024);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
#endif
	xtime = timerCall.Stop();
	CHECK_ERRORS(hipMemcpy(h_odata, d_odata, outputSize, hipMemcpyDeviceToHost));
	xmtime = timerMemory.Stop();
	hipFree(d_idata);
	hipFree(d_odata);
	printf("GPU Times : execution: %f, with memory: %f\n", xtime, xmtime);
	//auto res = vector<pair<int, int>>();
	auto res = ToPairVector(*h_odata);
	delete h_odata;
	return res;
}

vector<pair<int, int> > findPairsCPU(BitSequence<K> * sequence)
{
	BitSequence<L> *odata;
	odata = new BitSequence<L>();
	CudaTimer timerCall;
	timerCall.Start();
	checkSequencesCPU(sequence, odata);
	float xtime = timerCall.Stop();
	printf("CPU execution time: %f\n", xtime);
	auto res = ToPairVector(*odata);
	delete odata;
	return res;
}
