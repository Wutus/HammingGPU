#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm>

using namespace std;

//#define ITER_GPU //To omit watchdog on windows


#define CHECK_ERRORS(status) do{\
	if(hipSuccess != status) {\
		fprintf(stderr, "Cuda Error in %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(status));\
	}\
}while(0)

#define CHECK_ERRORS_FORMAT(status, format, ...) do{\
	if(hipSuccess != status) {\
		fprintf(stderr, "Cuda Error in %s:%d - %s", __FILE__, __LINE__, hipGetErrorString(status));\
		fprintf(stderr, format, __VA_ARGS__);\
		fprintf(stderr, "\n");\
	}\
}while(0)

#define BITS_IN_SEQUENCE 10000 //Number of bits in one sequence
#define INPUT_SEQUENCE_SIZE 100000ull //Number of sequences
#define COMPARISONS (((INPUT_SEQUENCE_SIZE*(INPUT_SEQUENCE_SIZE - 1)) / 2)) //Number of comparisons
#define MAX_BLOCKS 100000 //Number of maximum blocks per call
#define THREADS_IN_BLOCK 1024

template<unsigned long long k>
class BitSequence;
class CudaTimer;

class ResultArray
{
public:
	unsigned int **arr;
};

template<unsigned int N>
class HostResultArray : public ResultArray
{
public:

	HostResultArray()
	{
		arr = new unsigned int*[N - 1];

		for (int i = 0; i < N - 1; i++)
		{
			arr[i] = new unsigned int[ceil((i + 1) / 32.0)];
		}
	}

	~HostResultArray()
	{
		if (arr == nullptr)
			return;

		for (int i = 0; i < N - 1; i++)
		{
			delete[] arr[i];
		}

		delete[] arr;
	}

	/*HostResultArray<N>&& operator=(const HostResultArray<N> &h_result)
	{
		this->arr = h_result.arr;
		h_result.arr = nullptr;
	}*/

	HostResultArray<N>&& operator=(HostResultArray<N> &&h_result)
	{
		this->arr = h_result.arr;
		h_result.arr = nullptr;
	}

	HostResultArray(HostResultArray<N> &&h_result)
	{
		this->arr = h_result.arr;
		h_result.arr = nullptr;
	}

	char GetBit(unsigned int row, unsigned int col) const
	{
		return (char)(arr[row - 1][col / 32] >> (col % 32) & 1);
	}
};

template<unsigned int N>
class DeviceResultArray : public ResultArray
{
public:
	DeviceResultArray()
	{
		CHECK_ERRORS(hipMalloc(&arr, sizeof(unsigned int*)*(N - 1)));
		unsigned int* temp_arr[N - 1];
		for (int i = 0; i < N - 1; ++i)
		{
			CHECK_ERRORS(hipMalloc(&(temp_arr[i]), sizeof(unsigned int) * (ceil((i + 1) / 32.0))));
		}
		CHECK_ERRORS(hipMemcpy(arr, &(temp_arr[0]), sizeof(unsigned int*)*(N - 1), hipMemcpyHostToDevice));
	}

	~DeviceResultArray()
	{
		unsigned int **temp_arr[N - 1];
		CHECK_ERRORS(hipMemcpy(temp_arr, arr, sizeof(unsigned int*)*(N - 1), hipMemcpyDeviceToHost));
		for (int i = 0; i < N - 1; i++)
		{
			CHECK_ERRORS(hipFree(temp_arr[i]));
		}
		CHECK_ERRORS(hipFree(arr));
	}

	HostResultArray<N> ToHostArray()
	{
		HostResultArray<N> host;
		unsigned int *temp_arr[N - 1];
		CHECK_ERRORS(hipMemcpy(temp_arr, arr, sizeof(unsigned int*)*(N - 1), hipMemcpyDeviceToHost));
		for (int i = 0; i < N - 1; ++i)
		{
			CHECK_ERRORS(hipMemcpy(host.arr[i], temp_arr[i], sizeof(unsigned int) * (ceil((i + 1) / 32.0)), hipMemcpyDeviceToHost));
		}
		return host;
	}
};

__global__ void Hamming1GPU(BitSequence<BITS_IN_SEQUENCE> * d_sequence, BitSequence<COMPARISONS> *d_odata, unsigned long long offset = 0);
__host__ __device__ char compareSequences(BitSequence<BITS_IN_SEQUENCE> * sequence1, BitSequence<BITS_IN_SEQUENCE> * sequence2);
__host__ __device__ void k2ij(unsigned long long k, unsigned int * i, unsigned int  * j);
__host__ __device__ unsigned long long ij2k(unsigned int i, unsigned int j);
void Hamming1CPU(BitSequence<BITS_IN_SEQUENCE> * sequence, BitSequence<COMPARISONS> * odata);
void PrintComparison(const BitSequence<BITS_IN_SEQUENCE> & gpu_sequence, const BitSequence<BITS_IN_SEQUENCE> & cpu_sequence);
bool ComparePairs(const vector<pair<int, int> > & gpu_result, const vector<pair<int, int> > & cpu_result);

ostream & operator<<(ostream & out, BitSequence<BITS_IN_SEQUENCE> & sequence);
BitSequence<BITS_IN_SEQUENCE> * GenerateInput();
vector<pair<int, int> > ToPairVector(const BitSequence<COMPARISONS> & result_sequence);
void PrintAsMatrix(const BitSequence<COMPARISONS> & sequence, ostream & stream);

vector<pair<int, int> > FindPairsGPU(BitSequence<BITS_IN_SEQUENCE> * h_sequence);
vector<pair<int, int> > FindPairsGPU2(BitSequence<BITS_IN_SEQUENCE> * h_sequence);
vector<pair<int, int> > FindPairsCPU(BitSequence<BITS_IN_SEQUENCE> * sequence);
__host__ __device__ unsigned int* GetPointer(unsigned int **arr, unsigned int row, unsigned int col);
template<unsigned int N>
vector<pair<int, int> > ToPairVector(const HostResultArray<N> & result_array);

void PrintArray(BitSequence<BITS_IN_SEQUENCE> * arr);

int main()
{
	hipError_t cudaStatus;
	printf("Starting sequence generation...\n");
	BitSequence<BITS_IN_SEQUENCE>* sequence = GenerateInput();
	printf("Ended sequence generation!\n");

	auto gpuRes = FindPairsGPU2(sequence);
	auto cpuRes = FindPairsCPU(sequence);
	//PrintArray(sequence);
	ComparePairs(gpuRes, cpuRes);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

template<unsigned long long k>
class BitSequence
{
public:
	__host__ __device__ BitSequence()
	{

	}
	__host__ __device__ inline char GetBit(unsigned long long index) const
	{
		return (array[index / 64] >> (index % 64)) & 1;
	}
	__host__ __device__ inline void SetBit(unsigned long long index, char value)
	{
		array[index / 64] = (array[index / 64] & (~(1ull << (index % 64)))) | ((!!value) << (index % 64));
	}
	__host__ __device__ inline unsigned int *GetWord32(unsigned long long word_index)
	{
		return (((unsigned int*)array) + word_index);
	}
	__host__ __device__ inline unsigned long long *GetWord64(unsigned long long word_index)
	{
		return (array + word_index);
	}

	__host__ __device__ BitSequence(const BitSequence<k> & sequence)
	{
		memcpy(array, sequence.array, arSize * 8);
	}

	__host__ __device__ const BitSequence<k> & operator=(const BitSequence<k> & sequence)
	{
		memcpy(array, sequence.array, arSize * 8);
		return sequence;
	}
	static const unsigned long long arSize = (k + 63) / 64;
private:
	unsigned long long array[arSize];
};

__host__ __device__ char compareSequences(BitSequence<BITS_IN_SEQUENCE> * sequence1, BitSequence<BITS_IN_SEQUENCE> * sequence2)
{
	int diff = 0;
	for (int j = 0; j < (BITS_IN_SEQUENCE + 63) / 64; ++j)
	{
		unsigned long long int a, b, axorb;
		a = *(sequence1->GetWord64(j));
		b = *(sequence2->GetWord64(j));
		axorb = a ^ b;
		diff += axorb == 0 ? 0 : (axorb & (axorb - 1) ? 2 : 1);
		if (diff > 1)
		{
			return 0;
		}
	}
	return !!diff;
}

__host__ __device__ void k2ij(unsigned long long k, unsigned int * i, unsigned int  * j)
{
	*i = (unsigned int)ceil((0.5 * (-1 + sqrtl(1 + 8 * (k + 1)))));
	*j = (unsigned int)((k + 1) - 0.5 * (*i) * ((*i) - 1)) - 1;
}

__host__ __device__ unsigned long long ij2k(unsigned int i, unsigned int j)
{
	return ((unsigned long long)i) * (i - 1) / 2 + j;
}

void Hamming1CPU(BitSequence<BITS_IN_SEQUENCE> * sequence, BitSequence<COMPARISONS> * odata)
{
	unsigned long long numberOfComparisons = COMPARISONS;
	int i1 = 1, i2 = 0;
	for (unsigned long long k = 0; k < numberOfComparisons / 32; ++k)
	{
		unsigned int result = 0;
		for (int i = 0; i < 32; i++)
		{
			result |= (unsigned int)(compareSequences(sequence + i1, sequence + i2)) << i;
			++i2;
			if (i2 == i1)
			{
				++i1;
				i2 = 0;
			}
		}
		*(odata->GetWord32(k)) = result;
	}
	if (numberOfComparisons % 32)
	{
		unsigned int result = 0;
		for (int i = 0; i < numberOfComparisons % 32; i++)
		{
			result |= (unsigned int)(compareSequences(sequence + i1, sequence + i2)) << i;
			++i2;
			if (i2 == i1)
			{
				++i1;
				i2 = 0;
			}
		}
		*(odata->GetWord32(numberOfComparisons / 32)) = result;
	}
}

class CudaTimer
{
public:
	CudaTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		started = false;
	}

	~CudaTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		started = true;
		hipEventRecord(start);
		hipEventSynchronize(start);
	}

	float Stop()
	{
		if (!started)
			return -1.0f;
		float ms;
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		started = false;
		return ms;
	}
private:
	bool started;
	hipEvent_t start, stop;
};

void PrintComparison(const BitSequence<BITS_IN_SEQUENCE> & gpu_sequence, const BitSequence<BITS_IN_SEQUENCE> & cpu_sequence)
{
	for (unsigned long long i = 0; i < INPUT_SEQUENCE_SIZE*(INPUT_SEQUENCE_SIZE - 1) / 2; ++i)
	{
		if (cpu_sequence.GetBit(i) != gpu_sequence.GetBit(i))
		{
			unsigned int i1, i2;
			k2ij(i, &i1, &i2);
			cout << "Difference on comparison number " << i << " (" << i1 << ", " << i2 << ") GPU " << (short int)gpu_sequence.GetBit(i) << " CPU " << (short int)cpu_sequence.GetBit(i) << endl;
		}
	}
}

bool ComparePairs(const vector<pair<int, int> > & gpu_result, const vector<pair<int, int> > & cpu_result)
{
	unsigned long long gsize = gpu_result.size(), csize = cpu_result.size();
	unsigned long long n = gsize < csize ? gsize : csize;

	vector<pair<int, int> > gpu_res(gpu_result);
	vector<pair<int, int> > cpu_res(cpu_result);
	sort(gpu_res.begin(), gpu_res.end());
	sort(cpu_res.begin(), cpu_res.end());

	const vector<pair<int, int> > & gv = csize > gsize ? cpu_res : gpu_res;
	bool equal = true;

	if (gsize != csize)
	{
		cout << "Number of elements is not equal (GPU: " << gsize << ", CPU: " << csize << ") !" << endl;
		equal = false;
	}
	else
	{
		cout << "Number of elements are equal (GPU: " << gsize << ", CPU: " << csize << ")" << endl;
	}

	int i;
	for (i = 0; i < n; ++i)
	{
		if (gpu_res[i] != cpu_res[i])
		{
			cout << "Difference on " << i << ": GPU: (" << gpu_res[i].first << ", " << gpu_res[i].second << ") CPU: ("
				<< cpu_res[i].first << ", " << cpu_res[i].second << ")" << endl;
			equal = false;
		}
		else
		{
			//cout << "Pair " << i << ": GPU: (" << gpu_res[i].first << ", " << gpu_res[i].second << ") CPU: ("
			//		<< cpu_res[i].first << ", " << cpu_res[i].second << ")" << endl;

		}

	}
	if (csize != gsize)
	{
		cout << "Rest pairs on " << ((csize > gsize) ? "CPU" : "GPU") << ":" << endl;
		for (; i < gv.size(); ++i)
		{
			cout << "(" << gv[i].first << ", " << gv[i].second << ")" << endl;
		}
	}
	if (equal)
	{
		cout << "Results are the same" << endl;
	}
	return equal;
}

__global__ void Hamming1GPU(BitSequence<BITS_IN_SEQUENCE> * d_sequence, BitSequence<COMPARISONS> *d_odata, unsigned long long offset)
{
	unsigned long long i = threadIdx.x + blockIdx.x * blockDim.x + offset;
	unsigned int i1, i2;
	k2ij(i, &i1, &i2);
	i2 = compareSequences(d_sequence + i1, d_sequence + i2);
	__syncthreads();
	i1 = __ballot(~0, i2);
	*(d_odata->GetWord32(i / 32)) = i1;
}

ostream & operator<<(ostream & out, BitSequence<BITS_IN_SEQUENCE> & sequence)
{
	for (unsigned long long i = 0; i < BITS_IN_SEQUENCE; ++i)
	{
		out << (short int)sequence.GetBit(i);
	}
	return out;
}

BitSequence<BITS_IN_SEQUENCE> * GenerateInput()
{
	//dla 2019 blad na 1 bicie
	srand(2019);

	BitSequence<BITS_IN_SEQUENCE> * r = new BitSequence<BITS_IN_SEQUENCE>[INPUT_SEQUENCE_SIZE];

	memset(r, 0, sizeof(BitSequence<BITS_IN_SEQUENCE>)*INPUT_SEQUENCE_SIZE);

	for (int i = 0; i < INPUT_SEQUENCE_SIZE; i++)
	{
		*(r[i].GetWord32(0)) = i % 1024;
		/*for (int j = 0; j < BITS_IN_SEQUENCE / 32; j++)
		{
			*(r[i].GetWord32(j)) = rand() + rand()*RAND_MAX;
		}
		if(BITS_IN_SEQUENCE % 32)
			*(r[i].GetWord32(BITS_IN_SEQUENCE / 32)) = (rand() + rand()*RAND_MAX)%(1<<(BITS_IN_SEQUENCE%32));*/
	}
	return r;
}

vector<pair<int, int> > ToPairVector(const BitSequence<COMPARISONS> & result_sequence)
{
	vector<pair<int, int> > result;
	for (unsigned long long k = 0; k < COMPARISONS; k++)
	{
		if (result_sequence.GetBit(k))
		{
			unsigned int i, j;
			k2ij(k, &i, &j);
			result.push_back(make_pair(i, j));
		}
	}

	return result;
}

void PrintAsMatrix(const BitSequence<COMPARISONS> & sequence, ostream & stream)
{
	for (int i = 0; i < INPUT_SEQUENCE_SIZE; ++i)
	{
		for (int j = 0; j < INPUT_SEQUENCE_SIZE; ++j)
		{
			if (j <= i)
			{
				cout << "  ";
			}
			else
			{
				cout << (short int)sequence.GetBit(ij2k(i, j)) << " ";
			}
		}
		cout << endl;
	}
}

vector<pair<int, int> > FindPairsGPU(BitSequence<BITS_IN_SEQUENCE> * h_sequence)
{
	BitSequence<BITS_IN_SEQUENCE> *d_idata;
	BitSequence<COMPARISONS> *h_odata, *d_odata;
	CudaTimer timerCall, timerMemory;
	float xtime, xmtime;
	h_odata = new BitSequence<COMPARISONS>;
	unsigned long long inputSize = sizeof(BitSequence<BITS_IN_SEQUENCE>)* INPUT_SEQUENCE_SIZE, outputSize = sizeof(BitSequence<COMPARISONS>);
	timerMemory.Start();
	CHECK_ERRORS(hipMalloc(&d_idata, inputSize));
	CHECK_ERRORS(hipMalloc(&d_odata, outputSize));
	CHECK_ERRORS(hipMemcpy(d_idata, h_sequence, inputSize, hipMemcpyHostToDevice));
	CHECK_ERRORS(hipMemcpy(d_odata, h_odata, outputSize, hipMemcpyHostToDevice));
	timerCall.Start();
#ifdef ITER_GPU
	unsigned long long offset = 0;
	for (; offset + B * THREADS_IN_BLOCK < L; offset += B * THREADS_IN_BLOCK)
	{
		Hamming1GPU << < B, THREADS_IN_BLOCK >> > (d_idata, d_odata, offset);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if (L - offset >= THREADS_IN_BLOCK)
	{
		Hamming1GPU << < (int)((L - offset) / THREADS_IN_BLOCK), THREADS_IN_BLOCK >> > (d_idata, d_odata, offset);
		offset += (L - offset) * THREADS_IN_BLOCK;
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if ((L - offset) % THREADS_IN_BLOCK)
	{
		Hamming1GPU << < 1, (int)(L - offset) >> > (d_idata, d_odata, offset);
		offset += L - offset;
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	CHECK_ERRORS(hipDeviceSynchronize());
#else
	if (COMPARISONS >= THREADS_IN_BLOCK)
	{
		Hamming1GPU << < (int)(COMPARISONS / THREADS_IN_BLOCK), THREADS_IN_BLOCK >> > (d_idata, d_odata, 0);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if (COMPARISONS % THREADS_IN_BLOCK)
	{
		Hamming1GPU << < 1, COMPARISONS % THREADS_IN_BLOCK >> > (d_idata, d_odata, (COMPARISONS / THREADS_IN_BLOCK) * THREADS_IN_BLOCK);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
#endif
	xtime = timerCall.Stop();
	CHECK_ERRORS(hipMemcpy(h_odata, d_odata, outputSize, hipMemcpyDeviceToHost));
	xmtime = timerMemory.Stop();
	hipFree(d_idata);
	hipFree(d_odata);
	printf("GPU Times : execution: %f, with copying memory: %f\n", xtime, xmtime);

	vector<pair<int,int> > res = ToPairVector(*h_odata);
	delete h_odata;
	return res;
}

vector<pair<int, int> > FindPairsCPU(BitSequence<BITS_IN_SEQUENCE> * sequence)
{
	BitSequence<COMPARISONS> *odata;
	odata = new BitSequence<COMPARISONS>();
	CudaTimer timerCall;
	timerCall.Start();
	Hamming1CPU(sequence, odata);
	float xtime = timerCall.Stop();
	printf("CPU execution time: %f\n", xtime);
	vector<pair<int, int> > res = ToPairVector(*odata);
	delete odata;
	return res;
}

#define SEQUENCES_PER_CALL 15
#define THREADS_PER_BLOCK 1024

__global__ void Hamming2GPU(BitSequence<BITS_IN_SEQUENCE> *sequences, unsigned int **arr, unsigned int row_offset, unsigned int column_offset)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int seq_no = tid + column_offset;

	BitSequence<BITS_IN_SEQUENCE> s = *(sequences + seq_no);
	__shared__ BitSequence<BITS_IN_SEQUENCE> ar[SEQUENCES_PER_CALL];
	if (threadIdx.x < SEQUENCES_PER_CALL)
	{
		ar[threadIdx.x] = *(sequences + row_offset - threadIdx.x);
	}
	__syncthreads();
	for (int i = 0; i < SEQUENCES_PER_CALL; ++i)
	{
		char res = 0;
		unsigned int seq2_no = row_offset - i;
		//printf("Seq_no = %d, seq2_no = %d, tid= %d, blockIdx = %d, block_dim = %d, row_offset = %d\n", seq_no, seq2_no, threadIdx.x, blockIdx.x, blockDim.x, row_offset);
		if (seq2_no == 0)
			break;
		if (seq2_no >= INPUT_SEQUENCE_SIZE || seq_no >= INPUT_SEQUENCE_SIZE)
			return;
		if (seq2_no > seq_no)
		{
			//printf("Comparing %d with %d - line %d\n", seq_no, seq2_no, __LINE__);
			res = compareSequences(&s, &(ar[i]));
			//printf("%d and %d - %d\n", seq_no, seq2_no, (short int)res);
			/*if (res != 0)
				printf("%d and %d\n", seq_no, seq2_no);*/
		}
		__syncthreads();
		unsigned int b = __ballot(~0, res);

		if (seq2_no > seq_no)
		{
			//printf("Seq_no = %d, seq2_no = %d, b = %d\n", seq_no, seq2_no, b);
			//printf("%d\n", *(GetPointer(arr, seq2_no, seq_no)));
			*(GetPointer(arr, seq2_no, seq_no)) = b;
			//printf("b = %d, val = %d\n", b, *(GetPointer(arr, seq2_no, seq_no)));
		}
	}
}

vector<pair<int, int> > FindPairsGPU2(BitSequence<BITS_IN_SEQUENCE> * h_sequence)
{
	BitSequence<BITS_IN_SEQUENCE> *d_idata;
	DeviceResultArray<INPUT_SEQUENCE_SIZE> d_result;
	CudaTimer timerCall, timerMemory;
	float xtime, xmtime;
	unsigned long long inputSize = sizeof(BitSequence<BITS_IN_SEQUENCE>)* INPUT_SEQUENCE_SIZE;
	timerMemory.Start();
	CHECK_ERRORS(hipMalloc(&d_idata, inputSize));
	CHECK_ERRORS(hipMemcpy(d_idata, h_sequence, inputSize, hipMemcpyHostToDevice));
	timerCall.Start();

	for (int j = INPUT_SEQUENCE_SIZE - 1; j >= 0; j -= SEQUENCES_PER_CALL)
	{
		if (j >= THREADS_PER_BLOCK)
		{
			Hamming2GPU <<< j / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (d_idata, d_result.arr, j, 0);
			//CHECK_ERRORS(hipDeviceSynchronize());
		}
		if (j % THREADS_PER_BLOCK > 0)
		{
			Hamming2GPU <<< 1, j%THREADS_PER_BLOCK >>> (d_idata, d_result.arr, j, j - (j%THREADS_PER_BLOCK));
			//CHECK_ERRORS(hipDeviceSynchronize());
		}
	}
	HostResultArray<INPUT_SEQUENCE_SIZE> h_result(d_result.ToHostArray());
	xtime = timerCall.Stop();
	xmtime = timerMemory.Stop();
	hipFree(d_idata);
	printf("GPU Times : execution: %f, with memory: %f\n", xtime, xmtime);
	//vector<pair<int,int> > res = vector<pair<int, int>>();
	vector<pair<int, int> > res = ToPairVector(h_result);
	return res;
}

template<unsigned int N>
vector<pair<int, int> > ToPairVector(const HostResultArray<N> & result_array)
{
	vector<pair<int, int> > result;
	for (int i = 1; i < N; ++i)
	{
		for (int j = 0; j < i; ++j)
		{
			if (result_array.GetBit(i, j))
			{
				result.push_back(make_pair(i, j));
			}
		}
	}

	return result;
}

__host__ __device__ unsigned int* GetPointer(unsigned int **arr, unsigned int row, unsigned int col)
{
	return arr[row - 1] + col / 32;
}

void PrintArray(BitSequence<BITS_IN_SEQUENCE> * arr)
{
	for (int i = 0; i < INPUT_SEQUENCE_SIZE; ++i)
	{
		cout << arr[i] << endl;
	}
}