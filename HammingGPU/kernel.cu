#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <bitset>

#include <utility>
#include <vector>

#include "BitSequence.cu"

using namespace std;

#define CHECK_ERRORS(status) do{\
	if(hipSuccess != status) {\
		fprintf(stderr, "Cuda Error in %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(status));\
	}\
}while(0)

#define CHECK_ERRORS_FORMAT(status, format, ...) do{\
	if(hipSuccess != status) {\
		fprintf(stderr, "Cuda Error in %s:%d - %s", __FILE__, __LINE__, hipGetErrorString(status));\
		fprintf(stderr, format, __VA_ARGS__);\
		fprintf(stderr, "\n");\
	}\
}while(0)

template<unsigned long long K>
__host__ __device__ char compareSequences(BitSequence<K> * sequence1, BitSequence<K> * sequence2)
{
	int diff = 0;
	for (int j = 0; j < (K + 63) / 64; ++j)
	{
		unsigned long long int a, b, xor;
		a = *(sequence1->GetWord64(j));
		b = *(sequence2->GetWord64(j));
		xor = a ^ b;
		diff += xor == 0 ? 0 : (xor & (xor -1) ? 2 : 1);
		if (diff > 1)
		{
			return 0;
		}
	}
	return !!diff;
}
template<unsigned long long N, unsigned long long K>
void checkSequencesCPU(BitSequence<K> * sequence, void * odatav)
{
	BitSequence<N*(N * 1) / 2> *odata = (BitSequence<N*(N * 1) / 2> *)odatav;
	unsigned long long numberOfComparisons = N * (N - 1) / 2;
	for (unsigned long long k = 0; k < numberOfComparisons; k += 32)
	{
		unsigned int result = 0;
		for (int i = 0; i < 32; i++)
		{
			unsigned int i1, i2;
			k2ij(k + i, &i1, &i2);
			result |= (unsigned int)(compareSequences<K>(sequence + i1, sequence + i2)) << i;
		}
		*(odata->GetWord32(k / 32)) = result;
	}
}

__host__ __device__ inline double sqrtp(unsigned long long a)
{
	return 0.0f;
}

__host__ __device__ inline void k2ij(unsigned long long  k, unsigned int * i, unsigned int  * j)
{
	//adding 1 to k to skip first result
	*i = (unsigned int)(0.5 * (-1 + sqrtl(1 + 8 * (k + 1))));
	//decreasing 1 from j , as we start from 0 not 1
	*j = (unsigned int)((k + 1) - 0.5 * (*i) * ((*i) - 1)) - 1;
}

__host__ __device__ inline unsigned long long ij2k(unsigned int i, unsigned int j)
{
	return i * (i - 1) / 2 + j;
}

template<unsigned long long N, unsigned long long K>
__global__ void checkSequencesGPU(BitSequence<K> * d_sequence, BitSequence<N*(N - 1) / 2> *d_odata, unsigned long long offset = 0)
{
	unsigned long long i = threadIdx.x + blockIdx.x * 512 + offset;
	unsigned int i1, i2;
	k2ij(i, &i1, &i2);
	i2 = compareSequences<K>(d_sequence + i1, d_sequence + i2);
	i1 = __ballot_sync(~0, compareSequences<K>(d_sequence + i1, d_sequence + i2));
	*(d_odata->GetWord32(i/32)) = i1;
}

template<unsigned long long N, unsigned long long K>
__global__ void checkSequencesGPU2(BitSequence<K> * d_sequence, BitSequence<N*(N - 1) / 2> *d_odata, unsigned long long offset = 0)
{
	//unsigned long long i = (unsigned long long)threadIdx.x + (unsigned long long)512 * (unsigned long long)blockIdx.x + offset;
	unsigned long long i = threadIdx.x + blockIdx.x * 512 + offset;
	unsigned int i1, i2;
	//printf("%d\n", blockIdx.x);
	k2ij(i, &i1, &i2);
	/*if (ij2k(i1, i2) != i)
	{
		printf("Error! ij2k not giving the same as k2ij! (i = %d, j = %d, k = %d)", i1, i2, i);
		return;
	}*/
	//*((unsigned int*)(d_odata + i / 32 * 4)) = i1; = 0;
	//((unsigned int*)(d_odata + i / 32 * 4)) = i1; = __ballot_sync(~0, compareSequences<K>(d_sequence + i1, d_sequence + i2));
	/*i2 = compareSequences<K>(d_sequence + i1, d_sequence + i2);
	if(!(i%32))
		*(unsigned int*)(d_odata + i / 32 * 4) = __ballot_sync(~0, i2);*/
		//*(unsigned int*)(d_odata + i / 32 * 4) = i1;
		//printf("Tid %d, i1 %d, i2 %d, res %d, bs %d\n", i, i1, i2, res, bs);
}


class CudaTimer
{
public:
	CudaTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
		started = false;
	}

	~CudaTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		started = true;
		hipEventRecord(start);
	}

	float Stop()
	{
		if (!started)
			return -1.0f;
		float ms;
		hipEventRecord(stop);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ms, start, stop);
		started = false;
		return ms;
	}
private:
	bool started;
	hipEvent_t start, stop;
};

template<unsigned long long K>
void PrintComparison(const BitSequence<K> & gpu_sequence, const BitSequence<K> & cpu_sequence, unsigned long long N)
{
	for (unsigned long long i = 0; i < N*(N - 1) / 2; ++i)
	{
		if (cpu_sequence.GetBit(i) != gpu_sequence.GetBit(i))
		{
			unsigned int i1, i2;
			k2ij(i, &i1, &i2);
			cout << "Difference on comparison number " << i << " (" << i1 << ", " << i2 << ") GPU " << (short int)gpu_sequence.GetBit(i) << " CPU " << (short int)cpu_sequence.GetBit(i) << endl;
			unsigned int diff = 0;
			for (unsigned int j = 0; j < K; ++j)
			{
				diff += gpu_sequence[i1].GetBit(j) ^ cpu_sequence[i2].GetBit(j);
			}
			cout << "No of diffs: " << diff << endl;
		}
	}
}

bool ComparePairs(const vector<pair<int, int>> & gpu_result, const vector<pair<int, int>> & cpu_result)
{
	unsigned long long gsize = gpu_result.size(), csize = cpu_result.size();
	unsigned long long n = gsize < csize? gsize : csize;
	const vector<pair<int, int>> & lv = gsize < csize ? gpu_result : cpu_result;
	const vector<pair<int, int>> & gv = gsize < csize ? cpu_result : gpu_result;
	bool equal = true;

	if (gsize != csize)
	{
		cout << "Number of elements is not equal (GPU: " << gsize << ", CPU: " << csize << ") !" << endl;
		equal = false;
	}

	int i;
	for (i = 0; i < n; ++i)
	{
		if (gpu_result[i] != cpu_result[i])
		{
			cout << "Difference on " << i << ": GPU: (" << gpu_result[i].first << ", " << gpu_result[i].second << ") CPU: ("
				<< cpu_result[i].first << ", " << cpu_result[i].second << ")";
			equal = false;
		}
	}
	if (csize != gsize)
	{
		cout << "Rest pairs on " << ((csize < gsize) ? "CPU" : "GPU") << ":" << endl;
		for (; i < gv.size(); ++i)
		{
			cout << "(" << gv[i].first << ", " << gv[i].second << ")" << endl;
		}
	}
	return equal;
}

const unsigned long long K = 10000; //Number of bits in one sequence
const unsigned long long N = 100000; //Number of sequences
const unsigned long long L = (N*(N - 1)) / 2; //Number of comparisons
const unsigned int B = 100; //Number of maximum blocks per call

ostream & operator<<(ostream & out, BitSequence<K> & sequence)
{
	for (unsigned long long i = 0; i < K; ++i)
	{
		out << (short int)sequence.GetBit(i);
	}
	return out;
}

BitSequence<K> * Generate()
{
	srand(2018);

	BitSequence<K> * r = new BitSequence<K>[N];
	//memset(r, 0, sizeof(BitSequence<K>)*N);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < K/32; j++)
		{
			*(r[i].GetWord32(j)) = rand() + rand()*RAND_MAX;
		}
	}
	return r;
}

vector<pair<int, int>> ToPairVector(const BitSequence<L> & result_sequence)
{
	vector<pair<int, int>> result;
	for (unsigned long long k = 0; k < L; k++)
	{
		if (result_sequence.GetBit(k))
		{
			unsigned int i, j;
			k2ij(k, &i, &j);
			result.push_back(make_pair(i, j));
		}
	}

	return result;
}

void printAsMatrix(const BitSequence<L> & sequence, ostream & stream)
{
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			if (j <= i)
			{
				cout << "  ";
			}
			else
			{
				cout << (short int)sequence.GetBit(ij2k(i, j)) << " ";
			}
		}
		cout << endl;
	}
}

void printAsMatrix(const vector<std::pair<int, int>>, ostream & stream)
{
	/*for (int j = 0; j < N; ++j)
	{
		for (int i = 1; j < N; ++i)
		{
			if (j <= i)
			{
				cout << "  ";
			}
			else
			{
				cout << sequence.GetBit(ij2k(i, j)) + '0' << " ";
			}
		}
		cout << endl;
	}*/
}

vector<pair<int, int>> findPairsGPU(BitSequence<K> * h_sequence)
{
	BitSequence<K> *d_idata;
	BitSequence<L> *h_odata, *d_odata;
	CudaTimer timerCall, timerMemory;
	float xtime, xmtime;
	h_odata = new BitSequence<L>;
	unsigned long long inputSize = sizeof(BitSequence<K>)* N, outputSize = sizeof(BitSequence<L>);
	timerMemory.Start();
	CHECK_ERRORS(hipMalloc(&d_idata, inputSize));
	CHECK_ERRORS(hipMalloc(&d_odata, outputSize));
	CHECK_ERRORS(hipMemcpy(d_idata, h_sequence, inputSize, hipMemcpyHostToDevice));
	CHECK_ERRORS(hipMemcpy(d_odata, h_odata, outputSize, hipMemcpyHostToDevice));
	timerCall.Start();
	unsigned long long offset = 0;
	/*for (; offset + B * 1024 < L; offset += B * 1024)
	{
		checkSequencesGPU<N, K> <<< B, 1024 >>> (d_idata, d_odata, offset);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if (L - offset >= 1024)
	{
		checkSequencesGPU<N, K> <<< (int)((L - offset) / 1024), 1024 >>> (d_idata, d_odata, offset);
		offset += (L - offset) * 1024;
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if ((L - offset) % 1024)
	{
		checkSequencesGPU<N, K> <<< 1, (int)(L - offset) >>> (d_idata, d_odata, offset);
		offset += L - offset;
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	CHECK_ERRORS(hipDeviceSynchronize());*/
	if (L >= 1024)
	{
		checkSequencesGPU<N, K> <<< (int)(L/1024), 1024 >> > (d_idata, d_odata, 0);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	if (L % 1024)
	{
		checkSequencesGPU<N, K> <<< 1, L % 1024 >> > (d_idata, d_odata, (L/1024)*1024);
		CHECK_ERRORS(hipDeviceSynchronize());
	}
	xtime = timerCall.Stop();
	CHECK_ERRORS(hipMemcpy(h_odata, d_odata, outputSize, hipMemcpyDeviceToHost));
	xmtime = timerMemory.Stop();
	hipFree(d_idata);
	hipFree(d_odata);
	printf("GPU Times : execution: %f, with memory: %f\n", xtime, xmtime);
	auto res = vector<pair<int, int>>();
	//auto res = ToPairVector(*h_odata);
	delete h_odata;
	return res;
}

vector<pair<int, int>> findPairsCPU(BitSequence<K> * sequence)
{
	BitSequence<L> *odata;
	odata = new BitSequence<L>();
	checkSequencesCPU<N,K>(sequence, odata);
	auto res = ToPairVector(*odata);
	delete odata;
	return res;
}

int main()
{
	hipError_t cudaStatus;
	printf("Starting sequence generation...\n");
	BitSequence<K>* sequence = Generate();
	printf("Ended sequence generation!\n");

	auto gpuRes = findPairsGPU(sequence);
	//auto cpuRes = findPairsCPU(sequence);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

//vector<pair<int, int>> findPairs(BitSequence<K> * h_sequence)
//{
//	hipError_t cudaStatus;
//	BitSequence<K> *d_sequence;
//
//	cudaStatus = hipMalloc((void**)&d_sequence, sizeof(BitSequence<K>)*N);
//	if (cudaStatus != hipSuccess)
//	{
//		fprintf(stderr, "hipMalloc failed!  Do you have a CUDA-capable GPU installed?");
//	}
//
//	cudaStatus = hipMemcpy(d_sequence, h_sequence, sizeof(BitSequence<K>)*N, hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess)
//	{
//		fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
//	}
//	//Too big to keep on stack
//	BitSequence<L> *d_odata, *h_odata_p, *h_odata_p2;
//	h_odata_p = new BitSequence<L>;
//	h_odata_p2 = new BitSequence<L>;
//	BitSequence<L> & h_odata = *h_odata_p;
//	BitSequence<L> & h_odata2 = *h_odata_p2;
//	//printAsMatrix(h_odata, cout);
//	cudaStatus = hipMalloc((void**)&d_odata, sizeof(BitSequence<L>));
//	if (cudaStatus != hipSuccess)
//	{
//		fprintf(stderr, "cudaMallocfailed!  Do you have a CUDA-capable GPU installed?\n");
//	}
//	printf("Reserved %d under %d\n", sizeof(BitSequence<L>), d_odata);
//	cudaStatus = hipMemcpy(h_odata_p, d_odata, sizeof(h_odata), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "hipMemcpy no 1 failed! %s\n", err_str);
//	}
//	hipEvent_t start, stop; 
//	float time;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);
//
//	hipEventRecord( start, 0 );
//	printf("Starting counting on GPU...\n");
//	unsigned long long offset = 0;
//	unsigned long long nT = N * (N - 1) / 2;
//	/*for(unsigned long long i = 0; 1024*L*i < nT; ++i)
//	{
//		checkSequences<N, K> <<< L, 1024 >>> (d_sequence, d_odata, offset);
//		offset += L*1024;
//		printf("offset %ull\n", offset);
//		cudaStatus = hipDeviceSynchronize();
//		if (cudaStatus != hipSuccess)
//		{
//			const char *err_str = hipGetErrorString(cudaStatus);
//			fprintf(stderr, "kernelCall failed on offset %ull! %s\n", err_str, offset);
//		}
//	}
//	checkSequences<N, K> <<<(nT%L), 1024 >>> (d_sequence, d_odata, offset);
//	offset += (nT%L)*1024;
//	printf("offset %ull\n", offset);
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "kernelCall failed on offset %ull! %s\n", err_str, offset);
//	}
//	checkSequences<N, K> <<<1, nT - offset >>> (d_sequence, d_odata, offset);
//	offset = nT;
//	printf("offset %ull\n", offset);*/
//	printf("Gonna run %llu blocks (%d), for %llu comparisons\n", L / 512, L / 512 < (1 << 30) - 1, L);
//	checkSequences<N, K> <<< (unsigned int)(L/512), 512 >>> (d_sequence, d_odata, 0);
//
//	/*cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "hipGetLastError failed! %s\n", err_str);
//	}*/
//
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "kernelCall failed (%s) on offset %llu!\n", err_str, offset);
//	}
//
//	offset = (nT / (unsigned long long)512) * (unsigned long long)512;
//	printf("offset: %llu\n", offset);
//	//checkSequences<N, K> <<<1, (unsigned int)(nT%512) >> > (d_sequence, d_odata, offset);
//
//	/*cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "kernelCall failed on offset %ull! %s\n", err_str, offset);
//	}*/
//
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "hipGetLastError failed! %s\n", err_str);
//	}
//
//	hipEventRecord( stop, 0 );
//	hipEventSynchronize( stop );
//
//	hipEventElapsedTime( &time, start, stop );
//
//	hipEventDestroy( start );
//	hipEventDestroy( stop );
//    printf("GPU Processing time: %f (ms)\n", time);
//
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "hipGetLastError failed! %s\n", err_str);
//	}
//
//	printf("sizeof(h_odata): %d, d_odata %d\n", sizeof(h_odata), sizeof(*d_odata));
//	cudaStatus = hipMemcpy(h_odata_p, d_odata, sizeof(h_odata), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "hipMemcpy failed! %s\n", err_str);
//	}
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess)
//	{
//		const char *err_str = hipGetErrorString(cudaStatus);
//		fprintf(stderr, "hipDeviceSynchronize returned error code %s after launching addKernel!\nCurrent offset %llu", err_str, offset);
//	}
//	//printAsMatrix(h_odata, cout);
//
//	/*hipEventCreate(&start);
//	hipEventCreate(&stop);
//	hipEventRecord(start, 0);
//	hipEventSynchronize(start);
//	printf("Starting counting on CPU... \n");
//	checkSequencesCPU<N, K>(h_sequence, &h_odata2);
//	
//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&time, start, stop);
//
//	hipEventDestroy(start);
//	hipEventDestroy(stop);
//    printf("CPU Processing time: %f (ms)\n", time);
//	//printAsMatrix(h_odata2, cout);
//	cout << "Comparison: " << endl;
//	
//	//printAsMatrix(h_odata, cout);
//	
//	// ZWRACANIE TABLICY DW�JEK*/
//	vector<pair<int, int>> result;
//
//	//hipFree(d_sequence);
//	//hipFree(d_odata);
//	//delete h_odata_p;
//	//delete h_odata_p2;
//	/*for (unsigned long long i = 0; i < N*(N-1)/2; ++i)
//	{
//		unsigned long long i1, i2;
//		k2ij(i, &i1, &i2);
//		cout << i1 << " " << i2 << ": " << (short int)h_odata.GetBit(i) << endl;
//	}*/
//	return result;
//}

/*BitSequence<L> *CPUHamming(BitSequence<K> * h_sequence)
{
	BitSequence<L> *h_odata_p = new BitSequence<L>;
	BitSequence<L> & h_odata2 = *h_odata_p;

	hipEvent_t start, stop; float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipEventSynchronize(start);

	printf("Starting counting on CPU... \n");
	checkSequencesCPU<N, K>(h_sequence, &h_odata2);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("CPU Processing time: %f (ms)\n", time);
	//printAsMatrix(h_odata2, cout);

	return h_odata_p;
}*/